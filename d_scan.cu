#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CHECK.h"
#include "d_scan.h"
#include "config.h"  //defines THREADSPERBLOCK
#include "wrappers.h"


static __global__ void sumKernel(int *, int *, int);
static __global__ void sweepKernel(int *, int *) ;
static void cpuScan(int *, int);
static void exclusiveScan(int *, int, int);
__device__ void gpuPrintVec(const char * label, int * vector, int length);
void cpuPrintVec(const char * label, int * vector, int length);

/* d_scan
 * This function is a wrapper for the exclusive scan that is 
 * performed on the GPU. It uses hipMalloc to create an input/output
 * array on the GPU and copies the CPU array to the GPU array. 
 * It initializes the timing functions and then calls the 
 * exclusiveScan function to do the scan.
 * You should not modify this function.
 *
 * @param - output contains a pointer to the array to hold
 *          the output of the scan when complete
 * @param - input contains a pointer to the input to the scan 
 * @param - length is the size of the output array
 * @param - numEles is the number of elements to partition
 *          to a thread to complete the final sum for the scan
 */
float d_scan(int * output, int * input, int length, int numEles)
{
    int * d_output;
    float cpuMsecTime = -1;
    hipEvent_t start_cpu, stop_cpu;

    //THIS FUNCTION IS COMPLETE

    //To reduce the amount of time spent doing memory allocations,
    //create a single input/output array for GPU
    CHECK(hipMalloc((void **)&d_output, sizeof(int) * length));
    CHECK(hipMemcpy(d_output, input, length * sizeof(int), 
                     hipMemcpyHostToDevice));

    //start the timing
    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    CHECK(hipEventRecord(start_cpu));

    //do the scan and wait for all threads to complete
    exclusiveScan(d_output, length, numEles);
    hipDeviceSynchronize();

    //stop the timing
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));

    //copy the output of the GPU to the CPU array
    hipMemcpy(output, d_output, length * sizeof(int), hipMemcpyDeviceToHost);

    //release the space for the GPU array
    CHECK(hipFree(d_output));

    return cpuMsecTime;
}

/*
 * exclusiveScan
 * Launches two kernels to performs the exclusive scan on the GPU
 *
 * @param - d_output array that contains the input
 *          to use for the exclusive scan and holds
 *          the output of the exclusive scan
 * @param - length of array; always a power of 2
 * @param - numEles is the number of elements to partition
 *          to a thread to complete the scan; must be
 *          a power of 2 and less than or equal to
 *          MAXELES
 *          
*/
void exclusiveScan(int * d_output, int length, int numEles)
{
   if (length <= THREADSPERBLOCK)
   {
      //If the length is less than or equal to the block size then
      //do the scan on the CPU. Do this by implementing
      //and calling the cpuScan function. 
      //The result of this scan needs to be stored in
      //d_output.  Note d_output is a pointer to data
      //in GPU memory.  To use the CPU function, you'll need to copy
      //that data into CPU memory, call the CPU function,
      //and then copy the result back into GPU memory.  You can
      //test this function without implementing any of the other
      //code by running the program on vectors who are not
      //greater than THREADSPERBLOCK in length.  For example,
      // ./scan -s 9

      /* put the code to do what was described above here */

      return;
   } else
   {
      //1) Launch the sweepKernel
      //THREADSPERBLOCK is defined; Use length and THREADSPERBLOCK
      //to define the grid.
      //The kernel needs to be passed an array S
      //that will hold the values to be added to complete
      //the partial scan in d_output. The size of S will be
      //dependent upon the size of the grid.
      //You'll need to define and create S. The kernel is
      //provided; you just need to launch it.

      /* missing code goes here */

      //2) Don't continue on until all of the threads terminate
      CHECK(hipDeviceSynchronize());

      //3) Perform an exclusive scan on S (by calling exclusiveScan)
      //Isn't recursion great?
      //If S is 1 2 3 4 then exclusiveScan sets it to 0 1 3 6 

      /* missing code goes here */

      //4) Write and launch the sumKernel to add the elements of
      //S to elements of d_output. Use same grid dimensions that you defined
      //before, but set the number of threads in a block to
      //THREADSPERBLOCK/numEles.  This way each block i in this
      //kernel launch operates on the same elements as block i in
      //the sweepKernel kernel launch, but may use fewer threads
      //to do the work (depending upon the value of numEles).  

      /* missing code goes here */
   }
}


/*
 * sweepKernel
 * Performs an exclusive scan on the data on the d_output
 * array. In addition, one thread in each block will set an 
 * element in the sum array to the value that needs to be
 * added to the elements in the next section of d_output
 * to complete the scan.
 *
 * @param - d_output points to an array in the global memory
 *          that holds the input and will be modified to hold
 *          the output
 * @param - sum points to an array to hold the value to be 
 *          added to the section handled by blockIdx.x + 1
 *          in order to complete the scan
*/
__global__ void sweepKernel(int * d_output, int * sum)
{
   //THIS FUNCTION IS COMPLETE
   
   __syncthreads();
   int tid = threadIdx.x;
   int blkD = blockDim.x;
   int blkI = blockIdx.x;

   //d_input points to the section of the input to be
   //handled by this block
   int * d_input = d_output + blkI * blkD;
   __shared__ int shInput[THREADSPERBLOCK];

   //initialize the value in the sum array
   if (tid == (blkD >> 2) - 1)
   {
      sum[blkI] = d_input[blkD - 1];
   }

   //all threads participate in loading a
   //value into the shared memory
   shInput[tid] = d_input[tid];

   __syncthreads();
   int thid0Index = 0;
   int index;
   for (int i = 1; i < blkD; i<<=1)
   {
      thid0Index = thid0Index + i; 
      index = thid0Index + tid * 2 * i;
      if (index < blockDim.x) 
      {
         shInput[index] += shInput[index-i];
      }
      __syncthreads();
   }
  
   //set the last element in the section to 0 
   //before the next sweep
   if (tid == (blkD >> 2) - 1) shInput[blkD - 1] = 0;
   __syncthreads();  
   int i, j, topIndex, botIndex, tmp;
   for (j=1, i = blkD >> 1; i >= 1; i >>= 1, j <<= 1)
   {
      //first iteration thread 0 is active
      //second iteration threads 0, 1 are active
      //third iteration threads 0, 1, 2, 4
      if (tid < j)
      {
         topIndex = (tid + 1) * 2 * i - 1;
         botIndex = topIndex - i;
         tmp = shInput[botIndex];
         shInput[botIndex] = shInput[topIndex];
         shInput[topIndex] += tmp;
      }
      __syncthreads();
   }
   d_input[tid] = shInput[tid];
   //update sum using last element in the block
   if (tid == (blkD >> 2) - 1) sum[blkI] += shInput[blkD - 1];

   __syncthreads();
}

/*
 * sumKernel
 * Adds elements in sum to the elements in the d_output array.
 * The elements in the d_output array are sectioned into chunks
 * of size THREADSPERBLOCK.  sum[0] is added to the first chunk.
 * sum[1] added to the second block, etc.  The work is partitioned
 * among the threads in the block using cyclic partitioning.  Each thread
 * computes numElements results.
 * @param - sum points to an array of values to use to update
 *          d_output
 * @param - d_output points to the array of partially scanned
 *          values
*/
__global__ void sumKernel(int * sum, int * d_output, int numElements)
{
   //YOU NEED TO WRITE THIS
}

/* 
 * cpuScan
 * Performs an exclusive scan operation on the CPU.
 * @param - vector is a pointer to array of integers to
 *          scan
 * @param - length is the number of elements in an array
 * @modifies - vector array
*/
void cpuScan(int * vector, int length)
{
   //YOU NEED TO WRITE THIS
   //It isn't the same as the one in the book. That 
   //does an inclusive scan.

}

/* 
 * gpuPrintVec
 * Prints the contents a vector that is in the GPU memory, 10 elements
 * per line.  This can be used for debugging.
*/
__device__ void gpuPrintVec(const char * label, int * vector, int length)
{
    int i;
    printf("%s", label);
    for (i = 0; i < length; i++)
    {
        if ((i % 10) == 0) printf("\n%4d: ", i);
        printf("%3d ", vector[i]);
    }
    printf("\n");
}

/* 
 * cpuPrintVec
 * Prints the contents a vector that is in the CPU memory, 10 elements
 * per line.  This can be used for debugging.
*/
void cpuPrintVec(const char * label, int * vector, int length)
{
    int i;
    printf("%s", label);
    for (i = 0; i < length; i++)
    {
        if ((i % 10) == 0) printf("\n%4d: ", i);
        printf("%3d ", vector[i]);
    }
    printf("\n");
}
