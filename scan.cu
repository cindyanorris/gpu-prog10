#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "h_scan.h"
#include "d_scan.h"
#include "wrappers.h"
//config.h defines the number of threads in a block (THREADSPERBLOCK), 
//the minimum value used for calculating the vector size (MINVEC),
//and the maximum value used for calculating the vector size (MAXVEC).
#include "config.h"     

//prototypes for functions in this file
static void initVector(int * array, int length);
static void parseArgs(int argc, char * argv[], int *, int *);
static void compare(int * result1, int * result2, int n);
static void printUsage();
static int isPowerOfTwo(unsigned int n);
void printVector(int * vector, int vectorLen);

/*
   driver for the exclusive scan program.  
*/
int main(int argc, char * argv[])
{
    int vectorLen, numEles;
    //get the length of the vector and the number of elements
    //partitioned to threads to finish the GPU scan
    parseArgs(argc, argv, &vectorLen, &numEles);
    int * h_vector = (int *) Malloc(sizeof(int) * vectorLen);
    int * h_result = (int *) Malloc(sizeof(int) * vectorLen);
    int * d_result = (int *) Malloc(sizeof(int) * vectorLen);
    float h_time, d_time, speedup;

    //initialize vector 
    initVector(h_vector, vectorLen);
    //printVector(h_vector, vectorLen);
  
    printf("\nScan of vector of size: %d\n", vectorLen);
    printf("Number of sums computed by a thread: %d\n", numEles); 

    //perform the scan on the CPU
    h_time = h_scan(h_result, h_vector, vectorLen);
    //printVector(h_result, vectorLen);
    printf("\nTiming\n");
    printf("------\n");
    printf("CPU: \t\t%f msec\n", h_time);

    //perform the scan on the GPU 
    d_time = d_scan(d_result, h_vector, vectorLen, numEles);
    //printVector(d_result, vectorLen);

    //compare GPU and CPU results 
    compare(h_result, d_result, vectorLen);
    printf("GPU: \t\t%f msec\n", d_time);
    speedup = h_time/d_time;
    printf("Speedup: \t%f\n", speedup);

    free(h_result);
    free(d_result);
    free(h_vector);
}    

/* 
    parseArgs
    This function parses the command line arguments to get
    the vector length of the vector for the scan.
    It also sets the number of elements allocated 
    to a single thread to complete the final scan step. If 
    the vector length or number of elements value is invalid, 
    it prints usage information and exits.
    Inputs:
    argc - count of the number of command line arguments
    argv - array of command line arguments
    vectorLen - pointer to an int to be set to the vector length
    numEles - pointer to an int to be set to the number of elements
              that the GPU code will partition to each thread 
              when completing the final sum of the exclusive scan
*/
void parseArgs(int argc, char * argv[], int * vectorLen, int * numEles)
{
    int vlen = (1 << MINVEC);  //set vector length to default
    int vfactor;
    int i, numeles = 1; 
    for (i = 1; i < argc; i++)
    {
        if (strncmp(argv[i], "-s", 3) == 0)
        {
            //get value provided by user to calculate the vector size
            vfactor = atoi(argv[i+1]);
            if (vfactor < MINVEC || vfactor > MAXVEC)
            {
                printf("\nInvalid vector size: %d\n\n", vfactor);
                printUsage();
            }
            //calculate the vector size
            vlen = (1 << vfactor);
            i++;
        } else if (strncmp(argv[i], "-n", 3) == 0)
        {
            //get value provided by user for partitioning on GPU
            numeles = atoi(argv[i+1]);
            if (!isPowerOfTwo(numeles) || numeles > MAXELES)
            {
                printf("\nInvalid partition size: %d\n\n", numeles);
                printUsage();
            }
            i++;
        } else if (strncmp(argv[i], "-h", 3) == 0)
        {
            //display help info
            printUsage();
        } else
        {
            printf("\nInvalid option %s\n\n", argv[i]);
            printUsage();
        }
    }
    (*vectorLen) = vlen;
    (*numEles) = numeles;
}

/*
    printUsage
    prints usage information and exits
*/
void printUsage()
{
    printf("\nThis program performs an exclusive scan of a vector.\n"); 
    printf("The scan is performed on the CPU and the GPU. The program\n");
    printf("verifies the GPU results by comparing them to the CPU\n");
    printf("results and outputs the times it takes to perform each scan.\n");
    printf("usage: scan [-h] [-s <vector size>] [-n <partition size>]\n");
    printf("       [-h] print usage information\n");
    printf("       <vector size> size of randomly generated vector");
    printf(" is (1 << <vector size>).\n");
    printf("                   Min <vector size> is %d.\n", MINVEC);            
    printf("                   Max <vector size> is %d.\n", MAXVEC);            
    printf("                   Default is %d.\n", (1 << MINVEC));
    printf("       <partition size> number of sums computed by a thread");
    printf(" when doing the\n");
    printf("                   final sum to complete the scan.\n");
    printf("                   Must be a power of 2 that is less than");
    printf(" or equal to %d.\n", MAXELES);
    printf("                   Default is 1.\n");
    exit(EXIT_FAILURE);
}

/* 
    initVector
    Initializes an array of ints of size
    length to random values between 0 and 5. 
    Inputs:
    array - pointer to the array to initialize
    length - length of array
*/
void initVector(int * array, int length)
{
    int i;
    for (i = 0; i < length; i++)
    {
        array[i] = (rand() % 5);
    }
}

/*
    compare
    Compares the values in two vectors and outputs an
    error message and exits if the values do not match.
    result1, result2 - int vectors
    n - length of each vector
*/
void compare(int * result1, int * result2, int n)
{
    int i;
    for (i = 0; i < n; i++)
    {
        int diff = abs(result1[i] - result2[i]);
        if (diff != 0) 
        {
            printf("GPU scan does not match CPU scan.\n");
            printf("cpu result[%d]: %d, gpu: result[%d]: %d\n", 
                   i, result1[i], i, result2[i]);
            exit(EXIT_FAILURE);
        }
    }
}

/*
    printVector
    prints the contents of a vector, 10 elements per line
    vector - pointer to the vector
    vectorLen - length of vector
*/
void printVector(int * vector, int vectorLen)
{
    for (int i = 0; i < vectorLen; i++)
    {
        if ((i % 10) == 0)printf("\n%4d: ", i);
        printf("%3d ", vector[i]);
    }
    printf("\n");
}

/*
   isPowerOfTwo
   returns true if parameter is a power of 2 > 0.
*/ 
int isPowerOfTwo(unsigned int n)
{
    return (n != 0) && ((n & (n - 1)) == 0);
}
